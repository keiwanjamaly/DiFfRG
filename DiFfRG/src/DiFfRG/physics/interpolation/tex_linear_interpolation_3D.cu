// DiFfRG
#include <DiFfRG/discretization/grid/coordinates.hh>
#include <DiFfRG/physics/interpolation/tex_linear_interpolation_3D.hh>

namespace DiFfRG
{
  template <typename NT, typename Coordinates>
  TexLinearInterpolator3D<NT, Coordinates>::TexLinearInterpolator3D(const std::vector<NT> &data,
                                                                    const Coordinates &coordinates)
      : TexLinearInterpolator3D(data.data(), coordinates)
  {
    if (data.size() != coordinates.size()) throw std::runtime_error("Data and coordinates must have the same size");
  }

  template <typename NT, typename Coordinates>
  TexLinearInterpolator3D<NT, Coordinates>::TexLinearInterpolator3D(const Coordinates &coordinates)
      : TexLinearInterpolator3D(std::vector<NT>(coordinates.size(), NT(0)), coordinates)
  {
  }

  template <typename NT, typename Coordinates>
  TexLinearInterpolator3D<NT, Coordinates>::TexLinearInterpolator3D(const NT *in_data, const Coordinates &coordinates)
      : size(coordinates.size()), shape(coordinates.sizes()), coordinates(coordinates), m_data(nullptr),
        device_array(nullptr), m_data_AD(nullptr), device_array_AD(nullptr), owner(true)
  {
    if constexpr (std::is_same_v<float, ReturnType>) {
      // Copy input data and possibly cast to float
      m_data = std::shared_ptr<float[]>(new float[size]);
      for (uint i = 0; i < size; ++i)
        m_data[i] = static_cast<float>(in_data[i]);
#ifdef __HIPCC__
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D, precheck");
      // Allocate device array and copy data
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
      hipMalloc3DArray(&device_array, &channelDesc, make_hipExtent(shape[2], shape[1], shape[0]));
      hipDeviceSynchronize();
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D, hipMalloc3DArray");

      // Array creation
      hipMemcpy3DParms copyParams = {0};
      copyParams.srcPtr = make_hipPitchedPtr(m_data.get(), shape[2] * sizeof(float), shape[2], shape[1]);
      copyParams.dstArray = device_array;
      copyParams.extent = make_hipExtent(shape[2], shape[1], shape[0]);
      copyParams.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&copyParams);
      hipDeviceSynchronize();
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D, hipMemcpy3D");

      // Specify texture
      struct hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = device_array;

      // Specify texture object parameters
      struct hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModeLinear;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;

      // Create texture object
      hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL);
      hipDeviceSynchronize();
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D, hipCreateTextureObject");
#endif
    } else if constexpr (std::is_same_v<autodiff::real, ReturnType>) {
      // Copy input data and possibly cast to float
      m_data = std::shared_ptr<float[]>(new float[size]);
      m_data_AD = std::shared_ptr<float[]>(new float[size]);
      for (uint i = 0; i < size; ++i) {
        m_data[i] = static_cast<float>(val(in_data[i]));
        m_data_AD[i] = static_cast<float>(derivative(in_data[i]));
      }

#ifdef __HIPCC__
      // Allocate device array and copy data
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
      hipMalloc3DArray(&device_array, &channelDesc, make_hipExtent(shape[2], shape[1], shape[0]));
      hipMalloc3DArray(&device_array_AD, &channelDesc, make_hipExtent(shape[2], shape[1], shape[0]));
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D");

      // Array creation
      hipMemcpy3DParms copyParams = {0};
      copyParams.srcPtr = make_hipPitchedPtr(m_data.get(), shape[2] * sizeof(float), shape[2], shape[1]);
      copyParams.dstArray = device_array;
      copyParams.extent = make_hipExtent(shape[2], shape[1], shape[0]);
      copyParams.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&copyParams);
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D");
      hipMemcpy3DParms copyParams_AD = {0};
      copyParams_AD.srcPtr = make_hipPitchedPtr(m_data_AD.get(), shape[2] * sizeof(float), shape[2], shape[1]);
      copyParams_AD.dstArray = device_array_AD;
      copyParams_AD.extent = make_hipExtent(shape[2], shape[1], shape[0]);
      copyParams_AD.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&copyParams_AD);
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D");

      // Specify texture
      struct hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeArray;
      resDesc.res.array.array = device_array;

      struct hipResourceDesc resDesc_AD;
      memset(&resDesc_AD, 0, sizeof(resDesc_AD));
      resDesc_AD.resType = hipResourceTypeArray;
      resDesc_AD.res.array.array = device_array_AD;

      // Specify texture object parameters
      struct hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      texDesc.addressMode[0] = hipAddressModeClamp;
      texDesc.addressMode[1] = hipAddressModeClamp;
      texDesc.addressMode[2] = hipAddressModeClamp;
      texDesc.filterMode = hipFilterModeLinear;
      texDesc.readMode = hipReadModeElementType;
      texDesc.normalizedCoords = 0;

      // Create texture object
      hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL);
      hipCreateTextureObject(&texture_AD, &resDesc_AD, &texDesc, NULL);
      hipDeviceSynchronize();
      check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D");
#endif
    }
  }

  template <typename NT, typename Coordinates>
  TexLinearInterpolator3D<NT, Coordinates>::TexLinearInterpolator3D(
      const TexLinearInterpolator3D<NT, Coordinates> &other)
      : size(other.size), shape(other.shape), coordinates(other.coordinates), owner(false)
  {
    m_data = other.m_data;
    m_data_AD = other.m_data_AD;
#ifdef __HIPCC__
    device_array = other.device_array;
    texture = other.texture;
    device_array_AD = other.device_array_AD;
    texture_AD = other.texture_AD;
    check_cuda("TexLinearInterpolator3D::TexLinearInterpolator3D(const TexLinearInterpolator3D&)");
#endif
  }

  template <typename NT, typename Coordinates> TexLinearInterpolator3D<NT, Coordinates>::~TexLinearInterpolator3D()
  {
#ifdef __HIPCC__
    if (owner && m_data != nullptr) {
      hipDestroyTextureObject(texture);
      hipFreeArray(device_array);
      check_cuda("TexLinearInterpolator3D::~TexLinearInterpolator3D");
    }
    if (owner && m_data_AD != nullptr) {
      hipDestroyTextureObject(texture_AD);
      hipFreeArray(device_array_AD);
      check_cuda("TexLinearInterpolator3D::~TexLinearInterpolator3D");
    }
#endif
  }

  template <typename NT, typename Coordinates> void TexLinearInterpolator3D<NT, Coordinates>::update()
  {
    if (!owner) throw std::runtime_error("Cannot update data of non-owner interpolator");
#ifdef __HIPCC__
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(m_data.get(), shape[2] * sizeof(float), shape[2], shape[1]);
    copyParams.dstArray = device_array;
    copyParams.extent = make_hipExtent(shape[2], shape[1], shape[0]);
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    if constexpr (std::is_same_v<ReturnType, autodiff::real>) {
      hipMemcpy3DParms copyParams_AD = {0};
      copyParams_AD.srcPtr = make_hipPitchedPtr(m_data_AD.get(), shape[2] * sizeof(float), shape[2], shape[1]);
      copyParams_AD.dstArray = device_array_AD;
      copyParams_AD.extent = make_hipExtent(shape[2], shape[1], shape[0]);
      copyParams_AD.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&copyParams_AD);
    }

    hipDeviceSynchronize();
    check_cuda("TexLinearInterpolator3D::update");
#endif
  }

  template <typename NT, typename Coordinates> float *TexLinearInterpolator3D<NT, Coordinates>::data() const
  {
    return m_data.get();
  }

  template <typename NT, typename Coordinates> float *TexLinearInterpolator3D<NT, Coordinates>::data_AD() const
  {
    return m_data_AD.get();
  }

  template <typename NT, typename Coordinates>
  typename TexLinearInterpolator3D<NT, Coordinates>::ReturnType &
  TexLinearInterpolator3D<NT, Coordinates>::operator[](const uint i)
  {
    if constexpr (std::is_same_v<ReturnType, autodiff::real>)
      throw std::runtime_error("Cannot access autodiff::real data directly");
    else if constexpr (std::is_same_v<ReturnType, float>)
      return m_data[i];
  }

  template <typename NT, typename Coordinates>
  const typename TexLinearInterpolator3D<NT, Coordinates>::ReturnType &
  TexLinearInterpolator3D<NT, Coordinates>::operator[](const uint i) const
  {
    if constexpr (std::is_same_v<ReturnType, autodiff::real>)
      throw std::runtime_error("Cannot access autodiff::real data directly");
    else if constexpr (std::is_same_v<ReturnType, float>)
      return m_data[i];
  }

  template class TexLinearInterpolator3D<
      double,
      CoordinatePackND<LogarithmicCoordinates1D<float>, LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class TexLinearInterpolator3D<
      float, CoordinatePackND<LogarithmicCoordinates1D<float>, LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class TexLinearInterpolator3D<
      autodiff::real,
      CoordinatePackND<LogarithmicCoordinates1D<float>, LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;

  template class TexLinearInterpolator3D<
      double,
      CoordinatePackND<LogarithmicCoordinates1D<float>, LogarithmicCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class TexLinearInterpolator3D<
      float,
      CoordinatePackND<LogarithmicCoordinates1D<float>, LogarithmicCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class TexLinearInterpolator3D<
      autodiff::real,
      CoordinatePackND<LogarithmicCoordinates1D<float>, LogarithmicCoordinates1D<float>, LinearCoordinates1D<float>>>;

  template class TexLinearInterpolator3D<
      double, CoordinatePackND<LinearCoordinates1D<float>, LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class TexLinearInterpolator3D<
      float, CoordinatePackND<LinearCoordinates1D<float>, LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class TexLinearInterpolator3D<
      autodiff::real,
      CoordinatePackND<LinearCoordinates1D<float>, LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;

  template class TexLinearInterpolator3D<
      double,
      CoordinatePackND<LogarithmicCoordinates1D<double>, LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
  template class TexLinearInterpolator3D<
      float,
      CoordinatePackND<LogarithmicCoordinates1D<double>, LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
  template class TexLinearInterpolator3D<
      autodiff::real,
      CoordinatePackND<LogarithmicCoordinates1D<double>, LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;

  template class TexLinearInterpolator3D<
      double, CoordinatePackND<LogarithmicCoordinates1D<double>, LogarithmicCoordinates1D<double>,
                               LinearCoordinates1D<double>>>;
  template class TexLinearInterpolator3D<
      float, CoordinatePackND<LogarithmicCoordinates1D<double>, LogarithmicCoordinates1D<double>,
                              LinearCoordinates1D<double>>>;
  template class TexLinearInterpolator3D<
      autodiff::real, CoordinatePackND<LogarithmicCoordinates1D<double>, LogarithmicCoordinates1D<double>,
                                       LinearCoordinates1D<double>>>;

  template class TexLinearInterpolator3D<
      double, CoordinatePackND<LinearCoordinates1D<double>, LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
  template class TexLinearInterpolator3D<
      float, CoordinatePackND<LinearCoordinates1D<double>, LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
  template class TexLinearInterpolator3D<
      autodiff::real,
      CoordinatePackND<LinearCoordinates1D<double>, LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
} // namespace DiFfRG
