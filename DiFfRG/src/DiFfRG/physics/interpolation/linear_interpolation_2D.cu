#include "hip/hip_runtime.h"
// DiFfRG
#include <DiFfRG/discretization/grid/coordinates.hh>
#include <DiFfRG/physics/interpolation/linear_interpolation_2D.hh>

namespace DiFfRG
{
  template <typename NT, typename Coordinates>
  LinearInterpolator2D<NT, Coordinates>::LinearInterpolator2D(const std::vector<NT> &data,
                                                              const Coordinates &coordinates)
      : LinearInterpolator2D(data.data(), coordinates)
  {
    if (data.size() != coordinates.size()) throw std::runtime_error("Data and coordinates must have the same size");
  }

  template <typename NT, typename Coordinates>
  LinearInterpolator2D<NT, Coordinates>::LinearInterpolator2D(const Coordinates &coordinates)
      : LinearInterpolator2D(std::vector<NT>(coordinates.size(), NT(0)), coordinates)
  {
  }

  template <typename NT, typename Coordinates>
  LinearInterpolator2D<NT, Coordinates>::LinearInterpolator2D(const NT *in_data, const Coordinates &coordinates)
      : size(coordinates.size()), shape(coordinates.sizes()), coordinates(coordinates), m_data(nullptr), owner(true)
  {
    // Copy input data
    m_data = std::shared_ptr<NT[]>(new NT[size]);
    for (uint i = 0; i < size; ++i)
      m_data[i] = NT(in_data[i]);
      // Create the thrust::device_vector
#ifdef __HIPCC__
    device_data = std::shared_ptr<thrust::device_vector<NT>>(new thrust::device_vector<NT>());
    device_data->resize(size);
    thrust::copy(m_data.get(), m_data.get() + size, device_data->begin());
    check_cuda("LinearInterpolator2D::LinearInterpolator2D");
    device_data_ptr = thrust::raw_pointer_cast(device_data->data());
#endif
  }

  template <typename NT, typename Coordinates>
  LinearInterpolator2D<NT, Coordinates>::LinearInterpolator2D(const LinearInterpolator2D<NT, Coordinates> &other)
      : size(other.size), coordinates(other.coordinates), owner(false), shape(other.shape)
  {
    m_data = other.m_data;
#ifdef __HIPCC__
    device_data_ptr = other.device_data_ptr;
#endif
  }

  template <typename NT, typename Coordinates> void LinearInterpolator2D<NT, Coordinates>::update()
  {
    if (!owner) throw std::runtime_error("Cannot update data of non-owner interpolator");

#ifdef __HIPCC__
    thrust::copy(m_data.get(), m_data.get() + size, device_data->begin());
    check_cuda("LinearInterpolator2D::update");
#endif
  }

  template <typename NT, typename Coordinates> NT *LinearInterpolator2D<NT, Coordinates>::data() const
  {
    return m_data.get();
  }

  template <typename NT, typename Coordinates>
  __forceinline__ __device__ __host__ NT LinearInterpolator2D<NT, Coordinates>::operator()(
      const typename Coordinates::ctype x, const typename Coordinates::ctype y) const
  {
#ifndef __CUDA_ARCH__
    using std::ceil;
    using std::floor;
    using std::max;
    using std::min;
#endif

    auto [idx_x, idx_y] = coordinates.backward(x, y);
    idx_x = max(static_cast<decltype(idx_x)>(0), min(idx_x, static_cast<decltype(idx_x)>(shape[0] - 1)));
    idx_y = max(static_cast<decltype(idx_y)>(0), min(idx_y, static_cast<decltype(idx_y)>(shape[1] - 1)));

#ifndef __CUDA_ARCH__
    const auto *d_ptr = m_data.get();
#else
    const auto *d_ptr = device_data_ptr;
#endif

    uint x1 = min(ceil(idx_x + static_cast<decltype(idx_x)>(1e-16)), static_cast<decltype(idx_x)>(shape[0] - 1));
    const auto x0 = x1 - 1;
    uint y1 = min(ceil(idx_y + static_cast<decltype(idx_y)>(1e-16)), static_cast<decltype(idx_y)>(shape[1] - 1));
    const auto y0 = y1 - 1;

    const auto corner00 = d_ptr[x0 * shape[1] + y0];
    const auto corner01 = d_ptr[x0 * shape[1] + y1];
    const auto corner10 = d_ptr[x1 * shape[1] + y0];
    const auto corner11 = d_ptr[x1 * shape[1] + y1];

    return corner00 * (x1 - idx_x) * (y1 - idx_y) + corner01 * (x1 - idx_x) * (idx_y - y0) +
           corner10 * (idx_x - x0) * (y1 - idx_y) + corner11 * (idx_x - x0) * (idx_y - y0);
  }

  template <typename NT, typename Coordinates> NT &LinearInterpolator2D<NT, Coordinates>::operator[](const uint i)
  {
    return m_data[i];
  }
  template <typename NT, typename Coordinates>
  const NT &LinearInterpolator2D<NT, Coordinates>::operator[](const uint i) const
  {
    return m_data[i];
  }

  template class LinearInterpolator2D<
      double, CoordinatePackND<LogarithmicCoordinates1D<float>, LogarithmicCoordinates1D<float>>>;
  template class LinearInterpolator2D<
      float, CoordinatePackND<LogarithmicCoordinates1D<float>, LogarithmicCoordinates1D<float>>>;
  template class LinearInterpolator2D<
      autodiff::real, CoordinatePackND<LogarithmicCoordinates1D<float>, LogarithmicCoordinates1D<float>>>;

  template class LinearInterpolator2D<double, CoordinatePackND<LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class LinearInterpolator2D<float, CoordinatePackND<LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;
  template class LinearInterpolator2D<autodiff::real,
                                      CoordinatePackND<LinearCoordinates1D<float>, LinearCoordinates1D<float>>>;

  template class LinearInterpolator2D<
      double, CoordinatePackND<LogarithmicCoordinates1D<double>, LogarithmicCoordinates1D<double>>>;
  template class LinearInterpolator2D<
      float, CoordinatePackND<LogarithmicCoordinates1D<double>, LogarithmicCoordinates1D<double>>>;
  template class LinearInterpolator2D<
      autodiff::real, CoordinatePackND<LogarithmicCoordinates1D<double>, LogarithmicCoordinates1D<double>>>;

  template class LinearInterpolator2D<double,
                                      CoordinatePackND<LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
  template class LinearInterpolator2D<float,
                                      CoordinatePackND<LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;
  template class LinearInterpolator2D<autodiff::real,
                                      CoordinatePackND<LinearCoordinates1D<double>, LinearCoordinates1D<double>>>;

} // namespace DiFfRG
